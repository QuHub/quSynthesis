#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaSequence.h"
#include "stdio.h"
#include "cuda_debug.h"


__global__ void cuSynthesizeKernel(CudaSequence *data);
void InitializeConstants();

void SynthesizeKernel(CudaSequence &seq)
{
  int bufferSizeBytes = seq.m_nVectorSizeBytes * seq.m_nSequences;
  CudaSequence *pcuSeq;

  // This is essential for Parallel Nsight debugging, since GPU1 is used to debug the
  // code, while GPU0 is used for the display.
  hipSetDevice(1);  

  int *ptr;
  CS( hipMalloc( (void**)&pcuSeq, sizeof(CudaSequence)) );
  CS( hipMalloc( (void**)&ptr, bufferSizeBytes) );
  CS( hipMalloc( (void**)&seq.m_cuIn, bufferSizeBytes) );
  CS( hipMalloc( (void**)&seq.m_cuOut, bufferSizeBytes) );
  CS( hipMalloc( (void**)&seq.m_cuTarget, seq.m_outputBlockSize) );
  CS( hipMalloc( (void**)&seq.m_cuOperation, seq.m_outputBlockSize) );
  CS( hipMalloc( (void**)&seq.m_cuControl, seq.m_outputBlockSize) );
  CS( hipMalloc( (void**)&seq.m_cuGates, seq.m_nSequences * sizeof(INT)) );

  // Copy memory block to CUDA device
  CS( hipMemcpy(seq.m_cuIn, seq.m_pIn, bufferSizeBytes, hipMemcpyHostToDevice) );
  CS( hipMemcpy(seq.m_cuOut, seq.m_pOut, bufferSizeBytes, hipMemcpyHostToDevice) );
  CS( hipMemcpy(pcuSeq, &seq, sizeof(seq), hipMemcpyHostToDevice) );

  InitializeConstants();
  cuSynthesizeKernel<<<1, 1>>>(pcuSeq);
  
  //// make the host block until the device is finished with foo
  //hipDeviceSynchronize();

  // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("My CUDA error: %s\n", hipGetErrorString(error));
  }

  hipMemcpy(seq.m_pTarget, seq.m_cuTarget, seq.m_outputBlockSize, hipMemcpyDeviceToHost);
  hipMemcpy(seq.m_pControl, seq.m_cuControl, seq.m_outputBlockSize, hipMemcpyDeviceToHost);
  hipMemcpy(seq.m_pOperation, seq.m_cuOperation, seq.m_outputBlockSize, hipMemcpyDeviceToHost);
  hipMemcpy(seq.m_pnGates, seq.m_cuGates, seq.m_nSequences * sizeof(INT), hipMemcpyDeviceToHost);

  // TODO: free up device memory
  hipFree(seq.m_cuIn);
  hipFree(seq.m_cuOut);
  hipFree(seq.m_cuTarget);
  hipFree(seq.m_cuOperation);
  hipFree(seq.m_cuControl);
  hipFree(seq.m_cuGates);
}