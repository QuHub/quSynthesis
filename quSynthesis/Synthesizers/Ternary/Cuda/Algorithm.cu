#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaSequence.h"

#include "stdio.h"

// Ternary Gates
int gTernaryOps[5][3]= 
{
  {2, 0, 1},      // 0: -1
  {1, 2, 0},      // 1: -2
  {1, 0, 2},      // 2: 01
  {2, 1, 0},      // 3: 02
  {0, 2, 1}       // 4: 12
};                

// Operation to use based on [input][output] values which are an index to the gates in the m_Op array above
// Example: 
//    Input = 2, Output = 1 => Gate 4 (From m_Op above would be Swap gate 12)
int gOpMap[3][3] =
{
  // Output    0, 1, 2     Input 
  {4, 2, 3}, // 0
  {2, 3, 4}, // 1
  {3, 4, 2}  // 2
};

// Bitmask two bits at a time for ternary operations.
int gBitMask[] = {3, 3<<2, 3<<4, 3<<6, 3<<8, 3<<10, 3<<12, 3<<14, 3<<16}; 

__device__ int *gcuBitMask;
__device__ int *gcuOpMap;
__device__ int *gcuTernaryOps;

__device__ void Process(int inTerm, int outTerm, int nBits, PINT gBitMask, PINT pTarget, PINT pControl, PINT pOperation);

void cudasafe( hipError_t error, char* file, int line)
{
  error = hipGetLastError();
   if(error!=hipSuccess) { 
     fprintf(stderr,"ERROR: File: %s(%d) : %s(%i)\n",file, line, hipGetErrorString(error), error);  
   }
}

#define CS(x) cudasafe(x, __FILE__, __LINE__)

__global__ void cuSynthesizeKernel(CudaSequence *data)
{
  CudaSequence seq = data[0];
  int index = blockIdx.x * seq.m_outputBlockSize;
  printf("nTerms: %d", seq.m_nTerms);

  for(int i=0; i<seq.m_nTerms; i++) {
    Process(seq.m_cuIn[index+i], 
            seq.m_cuOut[index+i], 
            seq.m_nBits,
            &seq.m_cuGates[index],
            &seq.m_cuTarget[index], 
            &seq.m_cuControl[index],
            &seq.m_cuOperation[index]
    );
  }

  //for(int i=0; i < 10; i++) {
  //  target[index+i] = 0x20;
  //  control[index+i] = index*100;
  //}

}

__device__ int Propagate(int outTerm);

#define BIT(x,i) ((x & gcuBitMask[i]) >> 2*i)
__device__ void Process(int inTerm, int outTerm, int nBits, PINT pnGates, PINT pTarget, PINT pControl, PINT pOperation)
{
  printf("I am in cuda");
  return;

  outTerm = Propagate(outTerm);

  // Process low to high transitions 
  //for(int dir=1; dir>-2; dir-=2) {
  //  for (int i=0; i < nBits; i++) {
  //    // Isloate bit (i) for processing
  //    int inBit  = (gcuBitMask[i] & inTerm);        // must be defined as signed int
  //    int outBit = (gcuBitMask[i] & outTerm);

  //    if ( dir * (inBit - outBit) > 0) {         // Difference? Yes!
  //      pTarget   [*pnGates] = i;                           // Save index of target bits
  //      pControl  [*pnGates] = ~gcuBitMask[i] & outTerm;      // For now, it is everything except target bits is a control bit
  //      pOperation[*pnGates++] = gcuOpMap[BIT(inTerm,i)][BIT(outTerm,i)];  // Find the appropriate operation. 
  //      outTerm = (~gcuBitMask[i] & outTerm) | (gcuBitMask[i] & inTerm);
  //    }
  //  }
  //}
}


__device__ int Propagate(int outTerm)
{

  return 0;
}

void InitializeConstants()
{
  CS( hipMalloc((void**)&gcuBitMask, sizeof(gBitMask)) );
  CS( hipMalloc((void**)&gcuTernaryOps, sizeof(gTernaryOps)) );
  CS( hipMalloc((void**)&gcuOpMap, sizeof(gOpMap)) );
  CS( hipMemcpy(gcuBitMask, gBitMask, sizeof(gBitMask), hipMemcpyHostToDevice) );
  CS( hipMemcpy(gcuTernaryOps, gTernaryOps, sizeof(gTernaryOps), hipMemcpyHostToDevice) );
  CS( hipMemcpy(gcuOpMap, gOpMap, sizeof(gOpMap), hipMemcpyHostToDevice) );
}


__global__ void cuPrintIt()
{
  // This requires setting the comupte mode 2.0 in the CUDA flags of the project configuration.
  // Properties -> CUDA C++ -> Device -> Code Generation= compute_20,sm_20 
  printf("What up: %d\n", blockIdx.x);
}

void PrintIt()
{

  cuPrintIt<<<1024,1>>>();
  hipDeviceReset();
}


void SynthesizeKernel(CudaSequence &seq)
{
  int bufferSizeBytes = seq.m_nVectorSizeBytes * seq.m_nSequences;
  CudaSequence *pcuSeq;

  InitializeConstants();

  // This is essential for Parallel Nsight debugging, since GPU1 is used to debug the
  // code, while GPU0 is used for the display.
  hipSetDevice(1);  

  int *ptr;
  CS( hipMalloc( (void**)&ptr, bufferSizeBytes) );
  CS( hipMalloc( (void**)&seq.m_cuIn, bufferSizeBytes) );
  CS( hipMalloc( (void**)&seq.m_cuOut, bufferSizeBytes) );
  CS( hipMalloc( (void**)&seq.m_cuTarget, seq.m_outputBlockSize) );
  CS( hipMalloc( (void**)&seq.m_cuOperation, seq.m_outputBlockSize) );
  CS( hipMalloc( (void**)&seq.m_cuControl, seq.m_outputBlockSize) );
  CS( hipMalloc( (void**)&seq.m_pnGates, seq.m_nSequences * sizeof(INT)) );
  CS( hipMalloc( (void**)&pcuSeq, sizeof(CudaSequence)) );

  // Copy memory block to CUDA device
  CS( hipMemcpy(seq.m_cuIn, seq.m_pIn, bufferSizeBytes, hipMemcpyHostToDevice) );
  CS( hipMemcpy(seq.m_cuOut, seq.m_pOut, bufferSizeBytes, hipMemcpyHostToDevice) );
  CS( hipMemcpy(pcuSeq, &seq, sizeof(seq), hipMemcpyHostToDevice) );

  cuSynthesizeKernel<<<1, 1>>>(pcuSeq);
  
  //// make the host block until the device is finished with foo
  //hipDeviceSynchronize();

  // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("My CUDA error: %s\n", hipGetErrorString(error));
  }

  hipMemcpy(seq.m_pTarget, seq.m_cuTarget, seq.m_outputBlockSize, hipMemcpyDeviceToHost);
  hipMemcpy(seq.m_pControl, seq.m_cuControl, seq.m_outputBlockSize, hipMemcpyDeviceToHost);
  hipMemcpy(seq.m_pOperation, seq.m_cuOperation, seq.m_outputBlockSize, hipMemcpyDeviceToHost);
  hipMemcpy(seq.m_pnGates, seq.m_cuGates, seq.m_nSequences * sizeof(INT), hipMemcpyDeviceToHost);

  hipFree(seq.m_cuIn);
  hipFree(seq.m_cuOut);
  hipFree(seq.m_cuTarget);
  hipFree(seq.m_cuOperation);
  hipFree(seq.m_cuControl);
  hipFree(seq.m_cuGates);
}