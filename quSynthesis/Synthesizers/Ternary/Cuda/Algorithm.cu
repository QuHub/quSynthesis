#include "hip/hip_runtime.h"


__global__ void cuSynthesizeKernel(const int *inp, const int *outp, int *target, int *control, int *operation)
{
  int index = blockIdx.x;
  target[index] = index;
  
  //for(int i=0; i < 10; i++) {
  //  target[index+i] = 0x20;
  //  control[index+i] = index*100;
  //}

}

void Process(int inTerm, int outTerm)
{
//  outTerm = Propogate(outTerm);

//  if (m_nGates > (m_nBufSize - THRESHOLD))  // make sure there is enough memory
//    Resize();

//  // Process low to high transitions 
//  for (int i=0; i < m_nBits; i++) 
//    ProcessBit(inTerm, outTerm, i, +1);

//  // Process high to low transitions 
//  for (int i=0; i < m_nBits; i++) 
//    ProcessBit(inTerm, outTerm, i, -1);
}

// Since each block executes on a core, we want to have a single thread per core
void SynthesizeKernel(const int *inp, const int *outp, int *target, int *control, int *operation, int size)
{
  cuSynthesizeKernel<<<size, 1>>>(inp, outp, target, control, operation);
}