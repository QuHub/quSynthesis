#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaSequence.h"
#include "stdio.h"
#include "cuda_debug.h"


#define BIT(x,i) ((x & gcuBitMask[i]) >> 2*i)

// Ternary Gates
int gTernaryOps[5][3]= 
{
  {2, 0, 1},      // 0: -1
  {1, 2, 0},      // 1: -2
  {1, 0, 2},      // 2: 01
  {2, 1, 0},      // 3: 02
  {0, 2, 1}       // 4: 12
};                

// Operation to use based on [input][output] values which are an index to the gates in the m_Op array above
// Example: 
//    Input = 2, Output = 1 => Gate 4 (From m_Op above would be Swap gate 12)
int gOpMap[3][3] =
{
  // Output    0, 1, 2     Input 
  {4, 2, 3}, // 0
  {2, 3, 4}, // 1
  {3, 4, 2}  // 2
};

// Bitmask two bits at a time for ternary operations.
int gBitMask[] = {3, 3<<2, 3<<4, 3<<6, 3<<8, 3<<10, 3<<12, 3<<14, 3<<16}; 
__device__ __constant__ int gcuBitMask[sizeof(gBitMask)];
__device__ __constant__ int gcuTernaryOps[5][3];
__device__ __constant__ int gcuOpMap[3][3];
__device__ void Process(int inTerm, int outTerm, int nBits, PINT gBitMask, PINT pTarget, PINT pControl, PINT pOperation);

void InitializeConstants()
{
  // Constants are scoped to a file, and cannot use extern..
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuBitMask), gBitMask, sizeof(gBitMask)) );
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuTernaryOps), gTernaryOps, sizeof(gTernaryOps)) );
  CS( hipMemcpyToSymbol(HIP_SYMBOL(gcuOpMap), gOpMap, sizeof(gOpMap)) );
}

__global__ void cuSynthesizeKernel(CudaSequence *data)
{
  CudaSequence seq = data[0];
  int index = blockIdx.x * seq.m_outputBlockSize;
  printf("nTerms: %d", seq.m_nTerms);
  seq.m_cuGates[index] = 0;

  for(int i=0; i<seq.m_nTerms; i++) {
    Process(seq.m_cuIn[index+i], 
            seq.m_cuOut[index+i], 
            seq.m_nBits,
            &seq.m_cuGates[index],
            &seq.m_cuTarget[index], 
            &seq.m_cuControl[index],
            &seq.m_cuOperation[index]
    );
  }

  //for(int i=0; i < 10; i++) {
  //  target[index+i] = 0x20;
  //  control[index+i] = index*100;
  //}

}

__device__ int Propagate(int outTerm);

__device__ void Process(int inTerm, int outTerm, int nBits, PINT pnGates, PINT pTarget, PINT pControl, PINT pOperation)
{
  printf("I am in cuda");

  outTerm = Propagate(outTerm);

  //  process low (output) to high (input) transitions first then high to low
  for(int dir=1; dir>-2; dir-=2) {
    for (int i=0; i < nBits; i++) {
      // Isloate bit (i) for processing
      int inBit  = (gcuBitMask[i] & inTerm);        // must be defined as signed int
      int outBit = (gcuBitMask[i] & outTerm);

      if ( dir * (inBit - outBit) > 0) {         // Difference? Yes!
        pTarget   [*pnGates] = i;                           // Save index of target bits
        pControl  [*pnGates] = ~gcuBitMask[i] & outTerm;      // For now, it is everything except target bits is a control bit
        pOperation[*pnGates++] = gcuOpMap[BIT(inTerm,i)][BIT(outTerm,i)];  // Find the appropriate operation. 
        outTerm = (~gcuBitMask[i] & outTerm) | (gcuBitMask[i] & inTerm);
      }
    }
  }
}


__device__ int Propagate(int outTerm)
{

  return 0;
}


