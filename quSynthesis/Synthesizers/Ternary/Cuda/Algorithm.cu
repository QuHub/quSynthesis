#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaSequence.h"

__global__ void cuSynthesizeKernel(CudaSequence *seq)
{
  int index = blockIdx.x;
  seq->m_cuTarget[index] = index;
  
  //for(int i=0; i < 10; i++) {
  //  target[index+i] = 0x20;
  //  control[index+i] = index*100;
  //}

}

void Process(int inTerm, int outTerm)
{
//  outTerm = Propogate(outTerm);

//  if (m_nGates > (m_nBufSize - THRESHOLD))  // make sure there is enough memory
//    Resize();

//  // Process low to high transitions 
//  for (int i=0; i < m_nBits; i++) 
//    ProcessBit(inTerm, outTerm, i, +1);

//  // Process high to low transitions 
//  for (int i=0; i < m_nBits; i++) 
//    ProcessBit(inTerm, outTerm, i, -1);
}

// Since each block executes on a core, we want to have a single thread per core
void SynthesizeKernel(const int *inp, const int *outp, int *target, int *control, int *operation, int size)
{
//  CudaSequence seq;

//  cuSynthesizeKernel<<<size, 1>>>(inp, outp, target, control, operation);
}

void SynthesizeKernel(CudaSequence &seq)
{
  int bufferSizeBytes = seq.m_nVectorSizeBytes * seq.m_nSequences;

  hipMalloc( (void**)&seq.m_cuIn, bufferSizeBytes);
  hipMalloc( (void**)&seq.m_cuOut, bufferSizeBytes);
  hipMalloc( (void**)&seq.m_cuTarget, seq.m_outputBlockSize);
  hipMalloc( (void**)&seq.m_cuOperation, seq.m_outputBlockSize);
  hipMalloc( (void**)&seq.m_cuControl, seq.m_outputBlockSize);

  // Copy memory block to CUDA device
  hipMemcpy(seq.m_cuIn, seq.m_pIn, bufferSizeBytes, hipMemcpyHostToDevice);
  hipMemcpy(seq.m_cuOut, seq.m_pOut, bufferSizeBytes, hipMemcpyHostToDevice);

  cuSynthesizeKernel<<<1024, 1>>>(&seq);

  if(seq.m_nGates * sizeof(INT) > seq.m_outputBlockSize)
    throw "Exceeded limit of buffer";

  seq.m_pTarget = new INT[seq.m_nGates];
  seq.m_pControl = new INT[seq.m_nGates];
  seq.m_pOperation = new INT[seq.m_nGates];

  int size = seq.m_nGates * sizeof(INT);
  hipMemcpy(seq.m_pTarget, seq.m_cuTarget, size, hipMemcpyDeviceToHost);
  hipMemcpy(seq.m_pControl, seq.m_cuControl, size, hipMemcpyDeviceToHost);
  hipMemcpy(seq.m_pOperation, seq.m_cuOperation, size, hipMemcpyDeviceToHost);

  hipFree(seq.m_cuIn);
  hipFree(seq.m_cuOut);
  hipFree(seq.m_cuTarget);
  hipFree(seq.m_cuOperation);
  hipFree(seq.m_cuControl);
}